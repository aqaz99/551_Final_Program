// File: stationary_seq.c
// Author: James Miners-Webb
// Description: This program is a seqential version of calculating how to hit a stationary target 
//               given initial velocity and distance to target. 
//              The method is to 'brute force' all possible angles that the 'cannon' can take from 0-90 degrees.
//              We will then find the function given our knowns and current angle and integrate it using Left Riemann Sum
//               to find the total distance and see if that is equivalent to our target distance. 
// Resources Used and Links:
//              - Results of output can be checked here for validity: https://amesweb.info/Physics/Projectile-Motion-Calculator.aspx


#include <hip/hip_runtime.h>
#include <stdio.h>
#include <math.h> // pow()
#include <assert.h>
#include <stdlib.h>

#define SIZE	90.0
#define THREADS 80.0 // This is total thread count, not threads per block
#define BLOCKS  20.0 // Try and stick to even numbers

// __global__ Lets compiler know we can run this function on GPU.
__global__ void calculateFiringSolutionInAngleRange(double targetDistance, double projectileVelocity, double initialProjectileHeight){

    // --- Establish work start and stop
    // Need to get work from where the last blocks last thread's work stops
	double work_per_thread = SIZE/(THREADS*BLOCKS);
	double threads_per_block = THREADS / BLOCKS;
	// printf("work per thread %f\n", work_per_thread);

	// Split up work
	double work_start = work_per_thread * (THREADS * blockIdx.x + threadIdx.x);
	double work_stop;

	// Last thread pick up remainder
	if(threadIdx.x == (THREADS-1) && blockIdx.x == BLOCKS-1){
		work_stop = SIZE;
	}else{
		work_stop = work_per_thread * (THREADS * blockIdx.x + threadIdx.x + 1);
	}
    // --- Establish work start and stop

    // Initial variables
    double stepSize = 100000;
    double deltax = targetDistance/stepSize;
    
    // Within .5 centimeters on the x axis
    double xToleranceToHit = 0.005;

    // Within .5 centimeters on the y axis
    double yToleranceToHit = 0.005;

    double maxProjectileDistance = 0.0;


    // printf("Block %d) Thread %d - Work start: %f, work stop:%f\n", blockIdx.x, threadIdx.x, work_start, work_stop);
    

    // Increase launch angle by a very small number as to cover many possible trajectories
    for(double angle=work_start; angle<work_stop; angle += .005){
        // printf("angle: %f\n", angle);
        // Variables for storing Riemann Sum values
        double x = 0;
        double projectileDistanceTraveled = 0;
        double angleInRadians = angle *  (M_PI / 180.0);


        while(1){
            // y = h + x * tan(α) - g * x² / (2 * V₀² * cos²(α)) // 4.9 because gravity is divided by 2
            double areaUnderSlice = initialProjectileHeight + x * tan(angleInRadians) - (9.8 * x * x / (2 * projectileVelocity * projectileVelocity * cos(angleInRadians) * cos(angleInRadians)));

            areaUnderSlice = areaUnderSlice * deltax;
            // If area under slice is negative, shot cannot reach or if we haven't found solution by the distance of the target
            if(areaUnderSlice < 0.0 || x > targetDistance){
                break;
            }

            // Total distance in x direction
            projectileDistanceTraveled += areaUnderSlice;

            // Check projectile elevation at given x, checking if less than zero, thus can't hit
            double projectileElevation = initialProjectileHeight + projectileDistanceTraveled * tan(angleInRadians) - (9.8 * projectileDistanceTraveled * projectileDistanceTraveled / (2 * projectileVelocity * projectileVelocity * cos(angleInRadians) * cos(angleInRadians)));
        
            // Y value is negative, can't hit target
            if(projectileElevation < 0.0){
                // printf("crossed x axis at %f\n", projectileDistanceTraveled);
                break;
            }

            // Candidate for hit, distance travelled by shot is almost equal to distance to target, now we need to check elevation
            if( fabs(targetDistance - projectileDistanceTraveled) <= xToleranceToHit){
                // printf("Difference between %f and %f: %f\n",targetDistance, projectileDistanceTraveled, fabs(targetDistance - projectileDistanceTraveled));
                // printf("Projectile elevation: %f\n",projectileElevation);
                // We can check projectile elevation by plugging into f() function with our distance as the x value now
                if( projectileElevation <= yToleranceToHit){
                    double travelTime = (projectileDistanceTraveled / (cos(angleInRadians) * projectileVelocity));

                    printf("-- Hit Target! --\nProjectile traveled %f meters in %f seconds with angle %f degrees.\n", projectileDistanceTraveled, travelTime, angle);
                    printf("Projectile elevation: %f\n",projectileElevation);
                    break;
                }
            }

            x += deltax;
        }
        if(projectileDistanceTraveled > maxProjectileDistance){
            maxProjectileDistance = projectileDistanceTraveled;
        }
        // printf("Final distance for %f = %f\n", angle, projectileDistanceTraveled);
    }
    // printf("Max projectile distance with initial velocity of %f is %f\n", projectileVelocity, maxProjectileDistance);
}

int main(int argc, char* argv[]){
    if(argc < 3){
        printf("Usage: ./stationary_seq target_distance initial_projectile_velocity\n");
        exit(1);
    }

    double targetDistance = atof(argv[1]);
    double projectileVelocity = atof(argv[2]);
    double initialProjectileHeight = 0;

    // cudaMallocManaged(&targetDistance, 2 * sizeof(double));
	// cudaMallocManaged(&projectileVelocity, 2 * sizeof(double));
	// cudaMallocManaged(&initialProjectileHeight, 2 * sizeof(double));


    // calculateFiringSolutionInAngleRange(targetDistance, projectileVelocity, initialProjectileHeight);
    calculateFiringSolutionInAngleRange <<<BLOCKS, THREADS>>>(targetDistance, projectileVelocity, initialProjectileHeight);

    // Like join? Or barrier?
	hipDeviceSynchronize();
    
    return 0;
}
