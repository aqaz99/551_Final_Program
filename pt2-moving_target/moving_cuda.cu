#include "hip/hip_runtime.h"

// https://www.omnicalculator.com/physics/range-projectile-motion

#include <stdio.h>
#include <math.h> // pow()
#include <assert.h>
#include <stdlib.h>

#define SIZE	90.0
#define THREADS 4.0 // This is total thread count, not threads per block
#define BLOCKS  1.0 // Try and stick to even numbers

typedef struct {
  double xPosition, yPosition, initialVelocity, initialHeight, firingAngle;
} ProjectileClass;

// __device__ means the function is called by GPU and is supposed to be executed on GPU
__device__ __host__ double predictedYValue(ProjectileClass* projectile, double x);
// __host__ means the functions is called by the CPU and is supposed to be executed on the CPU
__host__ void initProjectile(ProjectileClass* projectile, double initialVelocity, double initialHeight, double firingAngle);
__device__ __host__ double distanceGivenTime(double time, double angle, double projectileVelocity);
__host__ __device__ double timeGivenDistance(double distance, double angle, double projectileVelocity);
__host__ double calculateTotalDistance(ProjectileClass* projectile);
__host__ __device__ void printEquation(ProjectileClass* projectile);


// __global__ means the function is called by CPU and suppose to be executed on GPU
__global__ void calculateFiringSolutionInAngleRange(ProjectileClass* target, 
                                                    ProjectileClass* interceptor, 
                                                    double total_travel_time,
                                                    double targetX,
                                                    double targetY){

    // --- Establish work start and stop
    // Need to get work from where the last blocks last thread's work stops
	double work_per_thread = SIZE/(THREADS*BLOCKS);

	// Split up work
	double work_start = work_per_thread * (THREADS * blockIdx.x + threadIdx.x);
	double work_stop;

	// Last thread pick up remainder
	if(threadIdx.x == (THREADS-1) && blockIdx.x == BLOCKS-1){
		work_stop = SIZE;
	}else{
		work_stop = work_per_thread * (THREADS * blockIdx.x + threadIdx.x + 1);
	}

    int x;
    x = threadIdx.x;
    printf("%f\n", x);
    // --- Establish work start and stop
    // Within .05 meters or ~2 inches
    double xToleranceToHit = 0.05;

    // Within .05 meters or ~2 inches
    double yToleranceToHit = 0.05;

    // Lets say we want to hit our target at (travel time)/2 
    // 1. Find the x and y location of our target at that time
    // 2. Find an angle that an interceptor could be fired at that passes through that (x, y)
    // 3. Determine the time it would take for the interceptor to get to that location
    // 4. If the interceptor can reach that location in time, display when the interceptor needs to fire to hit the target

    // Initial Riemann sum variables
    double stepSize = 15000;
    double deltax = targetX/stepSize;

    for(double angle=work_start; angle<work_stop; angle+=.001){ // For all firing angles
        // Set new interceptor angle
        interceptor->firingAngle = angle;
        // Variables for storing Riemann Sum values
        double x = 0;
        double projectileDistanceTraveled = 0;
        while(1){
            // Should this x be dist traveled?
            double areaUnderSlice = predictedYValue(interceptor, x) * deltax;
            
            // If area under slice is negative, shot cannot reach or if we haven't found solution by the distance of the target
            if(areaUnderSlice < 0.0){
                break;
            }

            // Total distance in x direction
            projectileDistanceTraveled += areaUnderSlice;
            double projectileElevation = predictedYValue(interceptor, projectileDistanceTraveled);

            // Y value is negative, can't hit target
            if(projectileElevation < 0.0){
                break;
            }
            // Candidate for hit, distance travelled by shot is almost equal to distance to target, now we need to check elevation
            if( fabs(targetX - projectileDistanceTraveled) <= xToleranceToHit){
                // Get time it will take our projectile to hit the target
                double intercTimeToTarget = timeGivenDistance(targetX, interceptor->firingAngle, interceptor->initialVelocity);

                // Check to see if both y's are the same and if the time to target is positive, meaning the shot is possible. neg values are solutions but not in time
                if( fabs(projectileElevation - targetY) <= yToleranceToHit && ((total_travel_time/2) - intercTimeToTarget) > 0.0){
                    
                    // printf("-------Can hit target!------\n");
                    printf("- Angle: %f\n- Time to Target: %f seconds\n- Launch after: %f seconds\n", angle, intercTimeToTarget, (total_travel_time/2) - intercTimeToTarget);
                    // printf("- Target(x, y): (%f, %f)\n- Interceptor(x, y): (%f, %f)\n- ",targetX, targetY, projectileDistanceTraveled, projectileElevation);
                    // printEquation(interceptor);
                    // printf("----------------------------\n");
                    angle+=.1; // Increment angle by a good chunk as to not get a giant list of firing solutions
                    break;
                }
            }
            x += deltax;
        }
    }
}

int main(int argc, char* argv[]){
    // CL Arguments
    if(argc < 5){
        printf("Usage: ./moving_seq target_velocity target_initial_height target_firing_angle interceptor_velocity\n");
        exit(1);
    }

    double target_velocity = atof(argv[1]);
    double target_initial_height = atof(argv[2]);
    double target_firing_angle = atof(argv[3]);
    double interceptor_velocity = atof(argv[4]);

    // Init moving target with initial velocity of 35 and firing angle of 45
    ProjectileClass* target;
    hipMallocManaged(&target, SIZE * sizeof(ProjectileClass));
    initProjectile(target, target_velocity, target_initial_height, target_firing_angle);
    printEquation(target);

    // Init interceptor with initial velocity of user input and both initial height and angle to zero
    ProjectileClass* interceptor;
    hipMallocManaged(&interceptor, SIZE * sizeof(ProjectileClass));
    initProjectile(interceptor, interceptor_velocity, 0, 0);

    //print info about target projectile
    double total_distance_traveled = calculateTotalDistance(target);
    printf("Target final x position: %f\n", total_distance_traveled);
    double total_travel_time = timeGivenDistance(total_distance_traveled, target->firingAngle, target->initialVelocity);
    printf("Total travel time: %f\n", total_travel_time);


    // 1. Get (x, y) at total time / 2 
    printf("Attempting to hit target at %f\n", total_travel_time/2);
    double targetX = distanceGivenTime(total_travel_time/2, target->firingAngle, target->initialVelocity);
    double targetY = predictedYValue(target, targetX);
    printf("Targets position at %f seconds is (%f, %f)\n",total_travel_time/2, targetX, targetY);

    // Issue here is that I am executing code on the gpu, but calling functions on the cpu, need to remedy
    calculateFiringSolutionInAngleRange <<<2, 2>>>(target, interceptor, total_travel_time, targetX, targetY);

    // Like join from pthreads
	hipDeviceSynchronize();
    return 0;
}


// Equation taken from: https://www.omnicalculator.com/physics/trajectory-projectile-motion
__device__ double predictedYValue(ProjectileClass* projectile, double x){
    double angleInRadians = projectile->firingAngle *  (M_PI / 180.0);
    double y;
    double underTheDivision = (2 * projectile->initialVelocity * projectile->initialVelocity * cos(angleInRadians) * cos(angleInRadians));

    // printf("Under the division (IN METERS): %f\n", underTheDivision);
    
    // y = h + x * tan(α) - g * x² / (2 * V₀² * cos²(α)) // 4.9 because gravity is divided by 2
    y = projectile->initialHeight + x * tan(angleInRadians) - (9.8 * x * x / underTheDivision);

    return y;
}

__host__ void initProjectile(ProjectileClass* projectile, double initialVelocity, double initialHeight, double firingAngle){
    projectile->initialVelocity = initialVelocity;
    projectile->initialHeight = initialHeight;
    projectile->firingAngle = firingAngle;
}

__device__ double distanceGivenTime(double time, double angle, double projectileVelocity){
    double angleInRadians = angle *  (M_PI / 180.0);
    // distance = rate * time
    // Total distance traveled -> distance(x) = time * rate, where rate is cos(theta) * velocity
    return (time * cos(angleInRadians) * projectileVelocity);
}

__host__ double timeGivenDistance(double distance, double angle, double projectileVelocity){
    double angleInRadians = angle *  (M_PI / 180.0);
    // distance = rate * time
    // Total travel time -> Time = distance(x) / rate(cos(theta) * velocity)
    return (distance / (cos(angleInRadians) * projectileVelocity));
}

// https://www.omnicalculator.com/physics/range-projectile-motion
// This equation allows us to calculate the range of a projectile with height >= 0
// R = Vx * [Vy + √(Vy² + 2 * g * h)] / g
__host__ double calculateTotalDistance(ProjectileClass* projectile){

    // Get velocity in the x and y directions
    double angleInRadians = projectile->firingAngle *  (M_PI / 180.0);
    double Vx = projectile->initialVelocity * cos(angleInRadians);
    double Vy = projectile->initialVelocity * sin(angleInRadians);
    
    return (Vx * (Vy + sqrt(Vy * Vy + 2 * 9.8 * projectile->initialHeight)) / 9.8);
}

__host__ __device__ void printEquation(ProjectileClass* projectile){
    double angleInRadians = projectile->firingAngle *  (M_PI / 180.0);
    double underTheDivision = (2 * projectile->initialVelocity * projectile->initialVelocity * cos(angleInRadians) * cos(angleInRadians));
    printf("Function equation F(x) = %f + %fx - (9.8x^2) / %f\n",projectile->initialHeight, tan(angleInRadians), underTheDivision);
}